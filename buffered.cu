#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <iostream>
#include <algorithm> 
#include <time.h>

//sorting the elemens in decreasing order
__device__ void insertion_sort(float* knnqueue, int* knnqueueind, int qpitch, int k, int threadId){
	int n, o;
	float var;
	for (o = threadId; o < (k*qpitch)+threadId; o += qpitch){
		n = o;
		while (n > threadId){
			if (knnqueue[n] >knnqueue[n - qpitch]){

				var = knnqueue[n];
				knnqueue[n] = knnqueue[n - qpitch];
				knnqueue[n - qpitch] = var;
			}
			n -= qpitch;

		}
	}

}

__device__ void increasing_insertion_sort(float* knnqueue, int* knnqueueind, int qpitch, int k, int threadId){
	int n, o;
	float var;
	for (o = threadId; o < (k*qpitch)+threadId; o += qpitch){
		n = o;
		while (n > threadId){
			if (knnqueue[n]<knnqueue[n - qpitch]){

				var = knnqueue[n];
				knnqueue[n] = knnqueue[n - qpitch];
				knnqueue[n - qpitch] = var;
			}
			n -= qpitch;

		}
	}

}


__global__ void buffered_search(float *distlist, int *indlist, int listpitch,
	float *knnqueue, int *knnqueueind, int qpitch,int m,int k, int refnumber,
	int querynumber, float *dbuffer, int *ibuffer,int bsize,int bufpitch){
	int thx;
	thx = threadIdx.x + blockIdx.x*blockDim.x;
	int var2 = 0;
	int var3 = 0;
	volatile __shared__ int flag[128/32];
	int mval = m;
	int varpitch = qpitch;
	int buf_size_var = 0;
	int b, c;
	float var, var1;
	int move = mval / 2;
	int queueinit = 0;
	int j = 0;
	float locmax;
	int bufvar = thx;
	//int *flag = &flags[threadIdx.x / 32];
	if (thx < querynumber){
		
		//queue initialization
		if (queueinit == 0){
			while (j < k){
				knnqueue[j*qpitch + thx] = distlist[j*listpitch + thx];
				//printf("knnqueueu: %d	%f\n", thx, knnqueue[j*qpitch + thx]);
				j++;
			}
			insertion_sort(knnqueue, 0, qpitch, k, thx);
			queueinit = 1;

		}
		locmax = knnqueue[thx];

		//Checking the list of distances
		/*var3 = 0;
		for (int i = thx; i < (refnumber*listpitch) + thx; i += listpitch){
			if ((thx == 4444) && (distlist[i] <= 0.298632)){
				printf("LIST22: %d	%f\n", var3, distlist[i]);
				var3++;
			}
		}*/
		for (int i = k*listpitch + thx; i < refnumber*listpitch + thx; i += listpitch){
			
			//bufvar = thx;
			//flag[threadIdx.x / 32] = 0;
				if (distlist[i] <= locmax){
					dbuffer[bufvar] = distlist[i];
					bufvar += bufpitch;
					buf_size_var++;
				}
			if (buf_size_var == bsize){
				flag[threadIdx.x / 32] = 1;
				/*buf_size_var = 0;
				bufvar = thx;*/

			}
			if (flag[threadIdx.x / 32] == 1){
				flag[threadIdx.x / 32] = 0;
				bufvar = thx;
				increasing_insertion_sort(dbuffer, 0, bufpitch, buf_size_var, thx);
				/*if (thx == 0){
					var2 = 0;
					for (int y = thx; y < (buf_size_var*bufpitch) + thx; y += bufpitch){
						printf("bffeer : %d	%f\n", var2, dbuffer[y]);
						var2 += 1;
					}
					printf("\n");

				}*/
				//Insert from Buffer to Merge Queue
				locmax = knnqueue[thx];

				for (int insert = thx; insert < buf_size_var*bufpitch + thx; insert += bufpitch){
					mval = m;
					move = mval / 2;//reinitializing mval ,move to deal with the remaining elemnets in the list
					if (dbuffer[insert] <=locmax){
						knnqueue[thx] = dbuffer[insert];

						//insert to the first level m
						insertion_sort(knnqueue, 0, qpitch, m, thx);
						locmax = knnqueue[thx];
						/*if (thx == 0){
							var2 = 0;
							for (int y = thx; y < (k*qpitch) + thx; y += qpitch){
								printf("before bitnic : %d	%f\n", var2, knnqueue[y]);
								var2 += 1;
							}
							printf("\n");

						}*/
						while ((locmax < knnqueue[mval*qpitch + thx]) && (mval <= k)){
							//first bitonic sort step(two sorted list in decreasing order)
							for (int a = mval*qpitch + thx; (a < k*qpitch + thx) && (a < (2 * mval*qpitch) + thx); a += qpitch){
								if (knnqueue[a] > knnqueue[a - varpitch]){
									var = knnqueue[a];
									knnqueue[a] = knnqueue[a - varpitch];
									knnqueue[a - varpitch] = var;
								}
								else{
									break;//to finish the first bitonic step at the size of the previous level in the queue 
								}
								varpitch += 2 * qpitch;

							}//end of for
							//next bitonic sort steps
							while (move > 0){
								for (c = 0; (c < (k*qpitch) + thx) && (c < (mval * 2 * qpitch) + thx); c += move * 2 * qpitch){
									for (b = thx; (b < k*qpitch + thx) && (b < move*qpitch + thx); b += qpitch){
										if ((knnqueue[b + c] < knnqueue[b + (move*qpitch) + c]) && (b + (move*qpitch) + c < k*qpitch + thx)){
											var1 = knnqueue[b + c];
											knnqueue[b + c] = knnqueue[b + (move*qpitch) + c];
											knnqueue[b + (move*qpitch) + c] = var1;
										}

									}
								}

								move /= 2;
							}//end of while	
							//to compare with the next level in the queue
							locmax = knnqueue[mval*qpitch + thx];//to ensure that the level heads are in decreasing order.
							mval = mval * 2;
							move = mval / 2;
							varpitch = qpitch;
						}//the end of bitonic sort process for merging levels
					}
					else{
						break;
					}
					locmax = knnqueue[thx];//re_assigning the locmax to the head of the first level in the queue
				}//end for (int insert = thx; insert < bsize*bufpitch; insert += bufpitch)				
				buf_size_var = 0;

			}	
			locmax = knnqueue[thx];
		}//end of looping through distlist
	}
	if (thx ==0){
		var2 = 0;
		for (int y = thx; y < (k*qpitch) + thx; y += qpitch){
			printf("last result: %d	%f\n", var2, knnqueue[y]);
			var2 += 1;
		}
		printf("\n");

	}
}


int main(){
	//testing width=8000; h=32000 k=64
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int width =8192;//query points
	int height = 32768;//ref points 
	int k =1024;
	int bsize =16;
	//float var4[32] = { 20, 21, 16, 14, 22, 23, 24, 1,1,1,2,2,3,3,3,2,2,1,1,3,4,4,2,1,1,0,0,0,0,3,1,3 };
	size_t lpitch;
	size_t qpitch;
	size_t bufpitch;
	hipError_t val1,val2,val3;

	float *d_dbuffer;
	float *d_indistqueue;
	float *d_list;
	float *h_dbuffer = new float[width*bsize];
	float *qh_in = new float[width*k];
	float *l_in = new float[height*width];

	//allocate cpu memor
	float *h_out = (float *)malloc(width*k*sizeof(float));

	// generate the input array on the host/
	for (int i = 0; i < width*height; i++){
		l_in[i] = ((float)rand() / (float)RAND_MAX) * 10;
	}
	//l_in[i] = width*height - i;//should be used for testing bitonic sort
	//l_in = var4;
	val1 = hipMallocPitch((void **)&d_dbuffer, &bufpitch, width*sizeof(float), bsize);

	val2 = hipMallocPitch((void **)& d_indistqueue, &qpitch, width*sizeof(float), k);
	val3 = hipMallocPitch((void **)&d_list, &lpitch, width*sizeof(float), height);
	if (val1)
		printf("Memorypitch Error For buffer: %s\n", hipGetErrorString(val1));
	if (val2)
		printf("Memorypitch Error for queue: %s\n", hipGetErrorString(val1));
	if (val3)
		printf("Memorypitch Error for list: %s\n", hipGetErrorString(val1));

	// transfer the array to the GPU

	hipMemcpy2D(d_dbuffer, bufpitch, h_dbuffer, width*sizeof(float), width*sizeof(float), bsize, hipMemcpyHostToDevice);
	hipMemcpy2D(d_indistqueue, qpitch, qh_in, width*sizeof(float), width*sizeof(float), k, hipMemcpyHostToDevice);
	hipMemcpy2D(d_list, lpitch, l_in, width*sizeof(float), width*sizeof(float), height, hipMemcpyHostToDevice);
	// launch the kernel
	dim3 Grid(width / 128 + 1, 1, 1);
	dim3 threads(128, 1);
	hipEventRecord(start, 0);

	buffered_search <<<Grid, threads >> >(d_list, 0, lpitch / sizeof(float), d_indistqueue, 0, 
		qpitch/sizeof(float),8, k, height,width, d_dbuffer,0,bsize,bufpitch/sizeof(float));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	// make the host block until the device is finished with foo
	hipDeviceSynchronize();

	// check for error
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	// copy back the result array to the CPU
	hipMemcpy2D(h_out, width*sizeof(float), d_indistqueue, qpitch, width*sizeof(float), k, hipMemcpyDeviceToHost);
	//hipMemcpy(l_out, d_list, list_size*sizeof(int), hipMemcpyDeviceToHost);
	float milliseconds;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//
	/*for (int j = 0; j < k*width; j++)
	printf("dequeue: %d	%f\n", j, h_out[j]);
	*/
	printf("The required time:	%f\n", milliseconds / 1000);
	hipFree(d_indistqueue);
	hipFree(d_list);
	return 0;
}

