#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#include <iostream>
#include <algorithm> 
#include <time.h>

//__device__ void swap(int i, int j) {
//	float t;
//	float *a=new float[];
//	t = a[i];
//	a[i] = a[j];
//	a[j] = t;
//}


 __device__ void insertion_sort(float* knnqueue, int* knnqueueind, int qpitch, int k, int threadId){
	 int n, o;
	float var;
	 for (o = threadId; o < k*qpitch+threadId; o += qpitch){
		 n = o;
		 while (n > threadId){
			 if (knnqueue[n] >knnqueue[n - qpitch]){

				 var = knnqueue[n];
				 knnqueue[n] = knnqueue[n - qpitch];
				 knnqueue[n - qpitch] = var;
			 }
			 n -= qpitch;

		 }
	 }

 }
__global__ void selection_of_knn(float *distlist, int *indlist,int listpitch, int k, int numofpoint, int refdimofpoint,
	float *knnqueue, int *knnqueueind,int qpitch, int m,volatile int* result)
//the visibility of memory operations on the result variable is ensured by declaring it as volatile
//From Nvidia documentation.
{
	int thx;
	int queueinit = 0;
	int mval=m;
	float locmax;
	int	j = 0;
	int varpitch=qpitch;
	int b,c;//divfact for the dividing the sequence in the second bitonic sort step, 
	//initialization to 2 for the second stage of sorting as we devide the sequence into two lists and so on 
	float var, var1;
	//for bitonic diffrent stage sorting
	int move = mval / 2;

	thx = threadIdx.x + blockIdx.x*blockDim.x;
	if (thx < numofpoint){

//queue initialization
		if (queueinit == 0){
			while (j < k){
				knnqueue[j*qpitch + thx] = distlist[j*listpitch + thx];
				//printf("knnqueueu: %d	%f\n", thx, knnqueue[j*qpitch + thx]);
				j++;
			}
			insertion_sort(knnqueue, 0, qpitch, k, thx);
			queueinit = 1;
			
		}
		//insertion in the first level m
		//locmax is the first element in the queue
		locmax = knnqueue[thx];
		for (int i = k*listpitch + thx; i < refdimofpoint*listpitch+thx; i += listpitch){
			//locmax = knnqueue[thx];//re_assigning the locmax to the head of the first level in the queue
			mval = m;
			move = mval / 2;//reinitializing mval ,move to deal with the remaining elemnets in the list
			if (distlist[i] < locmax){
				knnqueue[thx] =  distlist[i];
				
				//insert to the first level m
				insertion_sort(knnqueue, 0, qpitch, m, thx);
				locmax = knnqueue[thx];
				while((locmax < knnqueue[mval*qpitch + thx])/*&&(mval<=k)*/){				
						//first bitonic sort step(two sorted list in decreasing order)
						for (int a = mval*qpitch + thx; (a<k*qpitch+thx)&&(a < (2 * mval*qpitch) + thx); a += qpitch){
							if (knnqueue[a] > knnqueue[a - varpitch]){
								var = knnqueue[a];
								knnqueue[a] = knnqueue[a - varpitch];
								knnqueue[a - varpitch] = var;
							}
							else{
								break;//to finish the first bitonic step at the size of the previous level in the queue 
							}
							varpitch += 2 * qpitch;

						}//end of for
							//next bitonic sort steps
							while (move >0){
								for (c = 0; (c<(k*qpitch) + thx) && (c < (mval * 2 * qpitch)+thx); c += move * 2 * qpitch){
									/*if (thx == 0)
									printf("move,localmax,mval,move*2,b:	%d	%f	%d	%d	\n", move, locmax, mval, move * 2);*/
									for (b = thx; (b<k*qpitch + thx) && (b < move*qpitch + thx); b += qpitch){
										if ((knnqueue[b + c] < knnqueue[b + (move*qpitch) + c]) && (b + (move*qpitch) + c<k*qpitch+thx)){
											var1 = knnqueue[b+c];
											knnqueue[b+c] = knnqueue[b + (move*qpitch)+c];
											knnqueue[b + (move*qpitch)+c] = var1;
										
											/*if (thx == 0){
												for (int u = thx; u < k*qpitch; u += qpitch)
													printf("next bitonic: %d	%f\n", thx, knnqueue[u]);
												printf("\n");
											}*/
										}

										/*if (knnqueue[b + mval*qpitch] < knnqueue[b + (mval + move)*qpitch]){
											var2 = knnqueue[b + mval*qpitch];
											knnqueue[b + mval*qpitch] = knnqueue[b + (mval + move)*qpitch];
											knnqueue[b + (mval + move)*qpitch] = var2;
										}*/
										
									}
								}
								
								move /= 2;
							}//end of while	
							//to compare with the next level in the queue
							locmax = knnqueue[mval*qpitch+thx];//to ensure that the level heads are in decreasing order.
						/*	if (thx==0)
							printf("locmax:	%f\n", locmax);*/
							mval =mval* 2;
							move = mval / 2;
							varpitch = qpitch;
					}//the end of bitonic sort process for merging levels
			}
			locmax = knnqueue[thx];//re_assigning the locmax to the head of the first level in the queue
		}
		
		//bitonic merge
		//if the head of the second level is less than that of the first one of size m
	
	}//end of thx<numofpoint	
	/*if (thx == 66){
		for (int y = thx; y < (k*qpitch)+thx; y += qpitch){
			printf("last result: %d	%f\n", var2, knnqueue[y]);
			var2+=1;
		}
		printf("\n");

	}*/
	}



int main(){
	//testing width=8000; h=32000 k=64
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int width =8192;//query points
	int height =32768;//ref points 
	int k =512;
	size_t lpitch;
	size_t qpitch;
	hipError_t val;

	float *d_indistqueue;
	float *d_list;
	float *l_in = new float[height*width];
	float *qh_in = new float[width*k];
	//allocate cpu memor
	float *h_out = (float *)malloc(width*k*sizeof(float));

	// generate the input array on the host/
	for (int i = 0; i < width*height; i++)
		l_in[i] = (float)rand() / (float)RAND_MAX;
		//l_in[i] = width*height- i;//should be used for testing bitonic sort
	//float l_in[20] = {20,21,16,14,22,23,24,8,30,32,0,5,4,3,2,1,0,6,2,7};
	val=hipMallocPitch((void **) & d_indistqueue, &qpitch, width*sizeof(float), k);
	val=hipMallocPitch((void **) &d_list, &lpitch, width*sizeof(float), height);
	if (val)
		printf("Memorypitch Error: %s\n", hipGetErrorString(val));


	// transfer the array to the GPU
	hipMemcpy2D(d_indistqueue, qpitch, qh_in, width*sizeof(float), width*sizeof(float), k, hipMemcpyHostToDevice);
	hipMemcpy2D(d_list, lpitch, l_in, width*sizeof(float), width*sizeof(float), height, hipMemcpyHostToDevice);
	// launch the kernel
	dim3 Grid(width / 512 + 1, 1, 1);
	dim3 threads(512, 1);
	hipEventRecord(start,0);
	selection_of_knn <<<Grid, threads>> >(d_list, 0,lpitch/sizeof(float), k, width, height, d_indistqueue, 0,qpitch/sizeof(float),8,0);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	//bitonic_finalize_col<<<1,1>>>(d_indistqueue,1,1,8);

	//
	// copy back the result array to the CPU
	hipMemcpy2D(h_out,width*sizeof(float), d_indistqueue,qpitch ,width*sizeof(float),k, hipMemcpyDeviceToHost);
	//hipMemcpy(l_out, d_list, list_size*sizeof(int), hipMemcpyDeviceToHost);
	float milliseconds;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//
	/*for (int j = 0; j < k*width; j++)
		printf("dequeue: %d	%f\n", j, h_out[j]);
*/
	printf("The required time:	%f\n", milliseconds / 1000);

	hipFree(d_indistqueue);
	hipFree(d_list);
	return 0;
}